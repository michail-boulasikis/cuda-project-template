#include "hip/hip_runtime.h"
#include <kernels/add_vec.cuh>

namespace gpu {

namespace detail {
__global__ void add_vec(float *__restrict__ res, float *__restrict__ const a,
                        float *__restrict__ const b, size_t n) {
  size_t i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < n) {
    res[i] = a[i] + b[i];
  }
}
} // namespace detail

void add_vec(float *r, const float *a, const float *b, size_t n) {
  float *d_r, *d_a, *d_b;
  hipMalloc(&d_r, n * sizeof(float));
  hipMalloc(&d_a, n * sizeof(float));
  hipMalloc(&d_b, n * sizeof(float));
  hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);
  const size_t block_size = 256;
  const size_t grid_size = (block_size + n - 1) / block_size;
  detail::add_vec<<<grid_size, block_size>>>(d_r, d_a, d_b, n);
  hipMemcpy(r, d_r, n * sizeof(float), hipMemcpyDeviceToHost);
  return;
}
} // namespace gpu
